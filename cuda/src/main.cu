#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <thrust/extrema.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include <hip/hip_runtime.h>

#include "opencv2/opencv.hpp"
#include "opencv2/highgui.hpp"
#include ""

#include "stdafx.h"


#define Block_x 32
#define Block_y 32

using namespace std;
using namespace cv;

__global__ void GetLaplaceNormal(float* r_nx, float* r_ny, float* r_nz,
                                 float* laplace, float* Z) {
  int v = blockDim.y * blockIdx.y + threadIdx.y;
  int u = blockDim.x * blockIdx.x + threadIdx.x;
  if ((u >= 1) && (u < umax - 1) && (v >= 1) && (v < vmax - 1)) {
    const int idx0 = v * umax + u;
    const int left_idx = idx0 - 1;
    const int right_idx = idx0 + 1;
    const int up_idx = (v - 1) * umax + u;
    const int down_idx = (v + 1) * umax + u;
    float gv = Z[down_idx] - Z[up_idx];
    float gu = Z[right_idx] - Z[left_idx];
    r_nx[idx0] = gu * fx;
    r_ny[idx0] = gv * fy;
    r_nz[idx0] = -(Z[idx0] + (v - vo) * gv + (u - uo) * gu);

    if (r_nz[idx0] > 0) {
      r_nx[idx0] = -r_nx[idx0];
      r_ny[idx0] = -r_ny[idx0];
      r_nz[idx0] = -r_nz[idx0];
    }
    

    laplace[idx0] = fabs(
        4 * Z[idx0] - (Z[left_idx] + Z[right_idx] + Z[up_idx] + Z[down_idx]));
//    if (laplace[idx0] < 0) {
//      laplace[idx0] = -laplace[idx0];
//    }
  }
}

__global__ void GetFinalNormal(float* r_nx, float* r_ny, float* r_nz,
                               float* laplace, float* nx, float* ny,
                               float* nz) {
  int v = blockDim.y * blockIdx.y + threadIdx.y;
  int u = blockDim.x * blockIdx.x + threadIdx.x;
  if ((u >= 1) && (u < umax - 1) && (v >= 1) && (v < vmax - 1)) {
    const int idx0 = v * umax + u;
    const int left_idx = idx0 - 1;
    const int right_idx = idx0 + 1;
    const int up_idx = (v - 1) * umax + u;
    const int down_idx = (v + 1) * umax + u;
    float min_value = min(min(min(laplace[left_idx], laplace[right_idx]),
                              min(laplace[up_idx], laplace[down_idx])),
                          laplace[idx0]);

    if (min_value == laplace[idx0]) {
      nx[idx0] = r_nx[idx0];
      ny[idx0] = r_ny[idx0];
      nz[idx0] = r_nz[idx0];
    }
    if (min_value == laplace[left_idx]) {
      nx[idx0] = r_nx[left_idx];
      ny[idx0] = r_ny[left_idx];
      nz[idx0] = r_nz[left_idx];
    }
    if (min_value == laplace[right_idx]) {
      nx[idx0] = r_nx[right_idx];
      ny[idx0] = r_ny[right_idx];
      nz[idx0] = r_nz[right_idx];
    }

    if (min_value ==laplace[up_idx]) {
      nx[idx0] = r_nx[up_idx];
      ny[idx0] = r_ny[up_idx];
      nz[idx0] = r_nz[up_idx];
    }

    if (min_value == laplace[down_idx]) {
      nx[idx0] = r_nx[down_idx];
      ny[idx0] = r_ny[down_idx];
      nz[idx0] = r_nz[down_idx];
    }
  }
}

void cal(float* Z, float* cpu_nx, float* cpu_ny, float* cpu_nz);

int main(int, char) {

  check_gpu_compute_capability();

  const int pixel_number = vmax * umax;
  const int float_memsize = sizeof(float) * pixel_number;

  float* x = (float*)calloc(pixel_number, sizeof(float));
  float* y = (float*)calloc(pixel_number, sizeof(float));
  float* z = (float*)calloc(pixel_number, sizeof(float));
  float* cpu_z = (float*)calloc(pixel_number, sizeof(float));

  load_data(1, cpu_z);

  double st = clock();

  for (int i = 0; i < 1000; ++ i) {
    cal(cpu_z, x, y, z);
  }
  std::cout << (clock() - st) / CLOCKS_PER_SEC << std::endl;

  vis(x, y, z);
}


void cal(float* cpu_z, float* cpu_nx, float* cpu_ny, float* cpu_nz){
  // std::cout << "copy1 " << std::endl;
  const int pixel_number = vmax * umax;
  dim3 threads = dim3(Block_x, Block_y);
  dim3 blocks = dim3(idivup(umax, threads.x), idivup(vmax, threads.y));

  //���涼��GPU�����õı���
  float* r_nx;       //= (float*)calloc(pixel_number, sizeof(float));
  float* r_ny;       //= (float*)calloc(pixel_number, sizeof(float));
  float* r_nz;       //= (float*)calloc(pixel_number, sizeof(float));
  float* z_laplace;  //= (float*)calloc(pixel_number, sizeof(float));
  float* nx;         //= (float*)calloc(pixel_number, sizeof(float));
  float* ny;         //= (float*)calloc(pixel_number, sizeof(float));
  float* nz;         //= (float*)calloc(pixel_number, sizeof(float));

  float* Z;
  ////�������CPU����
  //float* cpu_nx = (float*)calloc(pixel_number, sizeof(float));
  //float* cpu_ny = (float*)calloc(pixel_number, sizeof(float));
  //float* cpu_nz = (float*)calloc(pixel_number, sizeof(float));
  //float* cpu_z = (float*)calloc(pixel_number, sizeof(float));

  // std::cout << "copy  2" << std::endl;

  const int float_memsize = sizeof(float) * pixel_number;

  // std::cout << "copy  3" << std::endl;

  // std::cout << "copy  4" << std::endl;

  static int flag = 0;
  if (flag == 0) {
    hipMalloc((void**)&r_nx, float_memsize);
    hipMalloc((void**)&r_ny, float_memsize);
    hipMalloc((void**)&r_nz, float_memsize);
    hipMalloc((void**)&z_laplace, float_memsize);

    hipMalloc((void**)&nx, float_memsize);
    hipMalloc((void**)&ny, float_memsize);
    hipMalloc((void**)&nz, float_memsize);

    hipMalloc((void**)&Z, float_memsize);
    flag = 1;
  }
  // std::cout << "copy " << std::endl;

  //�������ϸ����ڴ�Z��GPU��
  hipMemcpy(Z, cpu_z, float_memsize, hipMemcpyHostToDevice);

  // ��һ������������laplace������z����ֵ

   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start, NULL);

  // std::cout << "@@" << std::endl;
  GetLaplaceNormal<<<blocks, threads>>>(r_nx, r_ny, r_nz, z_laplace, Z);
  hipDeviceSynchronize();
  GetFinalNormal<<<blocks, threads>>>(r_nx, r_ny, r_nz, z_laplace, nx, ny, nz);
  hipDeviceSynchronize();
  // normal_estimation_bg_median<<<blocks, threads>>>(
  //    nx_dev, ny_dev, nz_dev, Volume_dev, normalization, visualization);

   hipEventRecord(stop, NULL);
   hipEventSynchronize(stop);
   float msecTotal = 1.0f;
   hipEventElapsedTime(&msecTotal, start, stop);
   std::cout << "runtime: " << msecTotal << std::endl;

  //��gpu������Ū��CPU

  hipMemcpy(cpu_nx, nx, float_memsize, hipMemcpyDeviceToHost);
  hipMemcpy(cpu_ny, ny, float_memsize, hipMemcpyDeviceToHost);
  hipMemcpy(cpu_nz, nz, float_memsize, hipMemcpyDeviceToHost);

  hipMemcpy(cpu_nx, r_nx, float_memsize, hipMemcpyDeviceToHost);
  hipMemcpy(cpu_ny, r_ny, float_memsize, hipMemcpyDeviceToHost);
  hipMemcpy(cpu_nz, r_nz, float_memsize, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  return;
}

/*
int main2(int, char) {
  check_gpu_compute_capability();

  // Setting kernel and nz_filter types
  kernel_type kernel = SOBEL;
  nz_filter_type nz_filter = MEDIAN;
  normalization_type normalization = POS;
  visualization_type visualization = OPEN;

  float min_runtime = 100;
  float max_runtime = 0;

  // Setting parameters
  const int pixel_number = vmax * umax;

  // Create blocks and threads
  dim3 threads = dim3(Block_x, Block_y);
  dim3 blocks = dim3(idivup(umax, threads.x), idivup(vmax, threads.y));

  // compute memsize
  const int char_memsize = sizeof(char) * pixel_number;
  const int float_memsize = sizeof(float) * pixel_number;

  // declare eight arrays
  char* M = (char*)calloc(pixel_number, sizeof(char));
  float* D = (float*)calloc(pixel_number, sizeof(float));
  float* Z = (float*)calloc(pixel_number, sizeof(float));
  float* X = (float*)calloc(pixel_number, sizeof(float));
  float* Y = (float*)calloc(pixel_number, sizeof(float));
  float* nx = (float*)calloc(pixel_number, sizeof(float));
  float* ny = (float*)calloc(pixel_number, sizeof(float));
  float* nz = (float*)calloc(pixel_number, sizeof(float));

  cv::Mat M_mat(vmax, umax, CV_8U, M);
  cv::Mat D_mat(vmax, umax, CV_32F, D);
  cv::Mat X_mat(vmax, umax, CV_32F, X);
  cv::Mat Y_mat(vmax, umax, CV_32F, Y);
  cv::Mat Z_mat(vmax, umax, CV_32F, Z);
  cv::Mat nx_mat(vmax, umax, CV_32F, nx);
  cv::Mat ny_mat(vmax, umax, CV_32F, ny);
  cv::Mat nz_mat(vmax, umax, CV_32F, nz);

  // Bind X, Y, Z and D with texture memory;
  hipChannelFormatDesc desc_X = hipCreateChannelDesc<float>();
  hipChannelFormatDesc desc_Y = hipCreateChannelDesc<float>();
  hipChannelFormatDesc desc_Z = hipCreateChannelDesc<float>();
  hipChannelFormatDesc desc_D = hipCreateChannelDesc<float>();

  hipArray *X_texture, *Y_texture, *Z_texture, *D_texture;

  hipMallocArray(&X_texture, &desc_X, umax, vmax);
  hipMallocArray(&Y_texture, &desc_Y, umax, vmax);
  hipMallocArray(&Z_texture, &desc_Z, umax, vmax);
  hipMallocArray(&D_texture, &desc_D, umax, vmax);

  // Create four arrays to store nx, ny, nz and volume;
  float *nx_dev, *ny_dev, *nz_dev, *Volume_dev;

  hipMalloc((void**)&nx_dev, float_memsize);
  hipMalloc((void**)&ny_dev, float_memsize);
  hipMalloc((void**)&nz_dev, float_memsize);
  hipMalloc((void**)&Volume_dev, float_memsize * 9);

  for (int frm = 1; frm <= 2500; frm++) {
            load_data(
                    torusknot,
                    1,
                    X,
                    Y,
                    Z,
                    D,
                    M);

    hipMemcpyToArray(X_texture, 0, 0, X, float_memsize,
                      hipMemcpyHostToDevice);
    hipMemcpyToArray(Y_texture, 0, 0, Y, float_memsize,
                      hipMemcpyHostToDevice);
    hipMemcpyToArray(Z_texture, 0, 0, Z, float_memsize,
                      hipMemcpyHostToDevice);
    hipMemcpyToArray(D_texture, 0, 0, D, float_memsize,
                      hipMemcpyHostToDevice);

    hipBindTextureToArray(X_tex, X_texture, desc_X);
    hipBindTextureToArray(Y_tex, Y_texture, desc_Y);
    hipBindTextureToArray(Z_tex, Z_texture, desc_Z);
    hipBindTextureToArray(D_tex, D_texture, desc_D);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, NULL);

    normal_estimation_bg_median<<<blocks, threads>>>(
        nx_dev, ny_dev, nz_dev, Volume_dev, normalization, visualization);

    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    float msecTotal = 1.0f;
    hipEventElapsedTime(&msecTotal, start, stop);
    std::cout << "runtime: " << msecTotal << std::endl;

    if (msecTotal < min_runtime) {
      min_runtime = msecTotal;
    }
    if (msecTotal > max_runtime) {
      max_runtime = msecTotal;
    }

    hipMemcpy(nx, nx_dev, float_memsize, hipMemcpyDeviceToHost);
    hipMemcpy(ny, ny_dev, float_memsize, hipMemcpyDeviceToHost);
    hipMemcpy(nz, nz_dev, float_memsize, hipMemcpyDeviceToHost);

    cv::Mat vis_mat(vmax, umax, CV_16UC3);
    output_visualization(nx, ny, nz, vis_mat);

    std::cout << "finish" << endl;

    namedWindow("result", WINDOW_AUTOSIZE);
    imshow("result", vis_mat);
    waitKey(30);

    std::cout << frm << endl;
  }
  std::cout << std::endl
            << std::endl
            << "runtime: " << min_runtime << std::endl;
  return 0;
}
*/
